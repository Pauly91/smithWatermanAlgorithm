
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>


#define idealBlockSize 25

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__device__ int MAX2(int a, int b)
{
	if(a >= b)
		return a;
	else
		return b;
}


__device__ int MAX3(int a, int b, int c)
{
	if(a >= b)
	{
		if(a >= c)
			return a;
		else
			return c;
	}
	else
	{
		if(b >= c)
			return b;
		else
			return c;
	}
}

__device__ int MAXn(int * array, int thread_2D_pos_y, int thread_2D_pos_x, int dnaStrandSize, int length, int Ge)
{
	int max = 0;
	int value;
	int i;
	for (i = 1; i < length; ++i)
	{
		value = (array[thread_2D_pos_y * dnaStrandSize + length - i] - i * Ge);
		if(value > max)
			max = value; 
	}
	return max;
}


__global__ void alignmentFinder(char *dna, char *dnaCompare, int *E_, int *H_,int *F, int *matrixH, int dnaTestStrandSize, int dnaStrandSize)
{

// need to work on this


    const int2 thread_2D_pos = make_int2( blockIdx.x * blockDim.x + threadIdx.x,
                                            blockIdx.y * blockDim.y + threadIdx.y);

    const int thread_1D_pos = thread_2D_pos.y * dnaStrandSize + thread_2D_pos.x;
 	

	int i;
	int Gs = 8;
	int Ge = 1;
	int S;

	for (i = 0; i < dnaStrandSize; ++i) //  startinf from 1 to account for initial zero matrix
	{// l for compare dna count

		if(dna[i] == dnaCompare[thread_2D_pos.x])
			S = 5;
		else
			S = -3;

		//printf("dna:%c dnaCompare:%c Gs:%d Ge:%d S:%d\n",dna[k],dnaCompare[l],Gs,Ge,S);

		F[thread_1D_pos] = MAX2(F[(thread_2D_pos.y - 1) * dnaStrandSize + thread_2D_pos.x], matrixH[(thread_2D_pos.y - 1) * dnaStrandSize + thread_2D_pos.x] - Gs) - Ge;
		H_[thread_1D_pos] = MAX3(matrixH[(thread_2D_pos.y - 1) * dnaStrandSize + thread_2D_pos.x -1 ] + S, F[thread_2D_pos.y * dnaStrandSize + thread_2D_pos.x],0);
		E_[thread_1D_pos] = MAXn(H_,thread_2D_pos.y, thread_2D_pos.x, dnaStrandSize, i, Ge);
		matrixH[thread_1D_pos] = MAX2(H_[thread_1D_pos], E_[thread_1D_pos] - Gs);
	}			//printf("-->i:%d j:%d F:%d H_:%d E_:%d matrixH:%d \n",i,j,F[i][j],H_[i][j],E_[i][j],matrixH[i][j]);	
}

int main(int argc, char const *argv[])
{
	int i,j,k,l;
	int *matrixH = NULL;
	int *F = NULL;
	int *H_ = NULL;
	int *E_ = NULL;
	
	int dnaStrandSize;
	int dnaTestStrandSize;
	int fscanfReturn;

	int *d_matrixH = NULL;
	int *d_F = NULL;
	int *d_H_ = NULL;
	int *d_E_ = NULL;
	

	char *dna = NULL;
	char *dnaCompare = NULL;

	char *d_dna = NULL;
	char *d_dnaCompare = NULL;



	FILE *fp = NULL;

	if(!(fp = fopen(argv[1],"r")))
	{
		printf("%s is not open !! \n",argv[1]);
	}

	fscanfReturn = fscanf(fp,"%d",&dnaStrandSize);
	if(fscanfReturn < 0)
		printf("Scanning of Value Failed\n");
	fscanfReturn = fscanf(fp,"%d",&dnaTestStrandSize);
	if(fscanfReturn < 0)
		printf("Scanning of Value Failed\n");

	printf("%d \n",dnaStrandSize);
	printf("%d \n",dnaTestStrandSize);
	
	dna = (char *) malloc(dnaStrandSize * sizeof(char));
	dnaCompare = (char *) malloc(dnaTestStrandSize * sizeof(char));	

	F = (int *) calloc((dnaTestStrandSize + 1) * ( dnaStrandSize + 1) , sizeof(int));
	H_ = (int *) calloc((dnaTestStrandSize + 1) * ( dnaStrandSize + 1) , sizeof(int));
	E_ = (int *) calloc((dnaTestStrandSize + 1) * ( dnaStrandSize + 1) , sizeof(int));
	matrixH = (int *) calloc((dnaTestStrandSize + 1) * ( dnaStrandSize + 1) , sizeof(int));


	for (i = 0; i < dnaStrandSize; ++i)
	{
		fscanfReturn = fscanf(fp," %c ",&dna[i]);
		if(fscanfReturn < 0)
			printf("Scanning of Value Failed\n");
	}

	for (i = 0; i < dnaTestStrandSize; ++i)
	{
		fscanfReturn = fscanf(fp," %c ",&dnaCompare[i]);
		if(fscanfReturn < 0)
			printf("Scanning of Value Failed\n");	
	}
	fclose(fp);


	gpuErrchk(hipMalloc((void**)&d_E_, (dnaStrandSize + 1) * (dnaTestStrandSize + 1) * sizeof(int)));
	gpuErrchk(hipMalloc((void**)&d_H_ ,(dnaStrandSize + 1) * (dnaTestStrandSize + 1) * sizeof(int)));
	gpuErrchk(hipMalloc((void**)&d_F,(dnaStrandSize + 1) * (dnaTestStrandSize + 1) * sizeof(int)));
	gpuErrchk(hipMalloc((void**)&d_matrixH,(dnaStrandSize + 1) * (dnaTestStrandSize + 1) * sizeof(int)));

	gpuErrchk(hipMemset(d_E_, 0,(dnaStrandSize + 1) * (dnaTestStrandSize + 1) * sizeof(int)));
	gpuErrchk(hipMemset(d_H_, 0,(dnaStrandSize + 1) * (dnaTestStrandSize + 1) * sizeof(int)));
	gpuErrchk(hipMemset(d_F, 0,(dnaStrandSize + 1) * (dnaTestStrandSize + 1) * sizeof(int)));
	gpuErrchk(hipMemset(d_matrixH, 0,(dnaStrandSize + 1) * (dnaTestStrandSize + 1) * sizeof(int)));


	gpuErrchk(hipMalloc((void**)&d_dna, dnaStrandSize * sizeof(char)));
	gpuErrchk(hipMalloc((void**)&d_dnaCompare, dnaTestStrandSize * sizeof(char)));


	gpuErrchk(hipMemcpy(dna, d_dna, dnaTestStrandSize * sizeof(char), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(dnaCompare, d_dnaCompare, dnaTestStrandSize * sizeof(char), hipMemcpyHostToDevice));

	dim3 grid(dnaTestStrandSize/idealBlockSize, dnaStrandSize/idealBlockSize);
	dim3 block(idealBlockSize, idealBlockSize);

	alignmentFinder<<<grid, block>>>(d_dna, d_dnaCompare, d_E_, d_H_, d_F, d_matrixH, dnaTestStrandSize, dnaStrandSize);

	gpuErrchk(hipMemcpy(matrixH, d_matrixH, (dnaStrandSize + 1) * (dnaTestStrandSize + 1) * sizeof(int) , hipMemcpyDeviceToHost));


	for (i = 0; i < dnaTestStrandSize + 1; ++i)
	{
		for (j = 0; j < dnaStrandSize + 1; ++j)
		{
			printf("%d ",matrixH[i * dnaStrandSize + j]);
		}
		printf("\n");
	}


	return 0;
}